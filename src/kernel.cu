
#include <hip/hip_runtime.h>
namespace poyenc {

struct ByPass final {
  template <typename T>
  __device__ T operator()(T value) const {
    return value;
  }
};

struct Activate final {
  template <typename T>
  __device__ T operator()(T value) const {
    return (0 < value ? value : 0);
  }
};

// FIXME: hiprtc doesn't support non-type template argument for now, to mimic passing
//        std::size_t to this template, LPerBlock should be specialization of std::integral_constant<>
template <typename DataType, typename LPerBlock, typename UnaryFunction>
__global__ void kernel(DataType *input, std::size_t length, DataType *output) {
  for (std::size_t idx = 0; idx < length; ++idx) {
    *output++ = UnaryFunction{}(*input++);
  }
}
}  // namespace poyenc
